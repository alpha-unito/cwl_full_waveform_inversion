#include "stdlib.h"
#include "math.h"
#include "sys/time.h"
#include "stdio.h"

#include <hip/hip_runtime.h>



#define _POSIX_C_SOURCE 200809L
#define START(S) struct timeval start_ ## S , end_ ## S ; gettimeofday(&start_ ## S , NULL);
#define STOP(S,T) gettimeofday(&end_ ## S, NULL); T->S += (double)(end_ ## S .tv_sec-start_ ## S.tv_sec)+(double)(end_ ## S .tv_usec-start_ ## S .tv_usec)/1000000;


#define dampL0_Host(x,y) damp[(x)*y_stride1 + (y)]
#define gradL0_Host(x,y) grad[(x)*y_stride2 + (y)]
#define recL0_Host(time,p_rec) rec[(p_rec) + (time)*p_rec_stride0]
#define rec_coordsL0_Host(p_rec,d) rec_coords[(d) + (p_rec)*d_stride0]
#define uL0_Host(time,x,y) u[(time)*x_stride0 + (x)*y_stride0 + (y)]
#define vL0_Host(t,x,y) v[(t)*x_stride0 + (x)*y_stride0 + (y)]
#define vpL0_Host(x,y) vp[(x)*y_stride1 + (y)]


#define dampL0(x,y) damp_dev[(x)*y_stride1 + (y)]
#define gradL0(x,y) grad_dev[(x)*y_stride2 + (y)]
#define recL0(time,p_rec) rec_dev[(p_rec) + (time)*p_rec_stride0]
#define rec_coordsL0(p_rec,d) rec_coords_dev[(d) + (p_rec)*d_stride0]
#define uL0(time,x,y) u_dev[(time)*x_stride0 + (x)*y_stride0 + (y)]
#define vL0(t,x,y) v_dev[(t)*x_stride0 + (x)*y_stride0 + (y)]
#define vpL0(x,y) vp_dev[(x)*y_stride1 + (y)]


struct dataobj
{
  void * __restrict__ data;
  unsigned long * size;
  unsigned long * npsize;
  unsigned long * dsize;
  long  * hsize;
  long  * hofs;
  long  * oofs;
  void * dmap;
} ;

struct profiler
{
  double section0=0.0;
  double section1=0.0;
  double section2=0.0;
} ;


#define NTHX 8
#define NTHY 16
#define NTH 128


__global__ void first_section(float *__restrict__ vp_dev, 
                              float *__restrict__ v_dev, 
                              float *__restrict__ damp_dev, 
                              const float r2, 
                              const float r1, 
                              const long  x_m, 
                              const long  x_M, 
                              const long  y_m, 
                              const long  y_M, 
                              const long  x_stride0, 
                              const long  y_stride0, 
                              const long  y_stride1,
                              const long  t0,
                              const long  t1,
                              const long  t2){

    long  x = x_m + blockIdx.x * blockDim.x + threadIdx.x;
    long  y = y_m + blockIdx.y * blockDim.y + threadIdx.y;


    if (x <= x_M && y <= y_M) {
        float r3 = 1.0F/(vpL0(x + 2, y + 2)*vpL0(x + 2, y + 2));

        vL0(t1, x + 4, y + 4) = (r3*(-(r1*(-2.0F*vL0(t0, x + 4, y + 4)) + r1*vL0(t2, x + 4, y + 4))) \
                              + r2*dampL0(x + 2, y + 2)*vL0(t0, x + 4, y + 4) \
                              + 8.33333315e-4F*(-vL0(t0, x + 2, y + 4) - vL0(t0, x + 4, y + 2) - vL0(t0, x + 4, y + 6) - vL0(t0, x + 6, y + 4)) \
                              + 1.3333333e-2F*(vL0(t0, x + 3, y + 4) + vL0(t0, x + 4, y + 3) + vL0(t0, x + 4, y + 5) + vL0(t0, x + 5, y + 4)) \
                              - 4.99999989e-2F*vL0(t0, x + 4, y + 4))/(r3*r1 + r2*dampL0(x + 2, y + 2));

    }

}


__global__ void second_section(float* __restrict__ vp_dev, 
                              float* __restrict__ rec_coords_dev, 
                              float* __restrict__ rec_dev,
                              float* __restrict__ v_dev,
                              const long p_rec_stride0,
                              const long  y_stride1, 
                              const long  d_stride0, 
                              const long  x_m, 
                              const long  y_m, 
                              const long  x_M, 
                              const long  y_M, 
                              const float dt, 
                              const float o_x, 
                              const float o_y, 
                              const long  p_rec_m, 
                              const long  p_rec_M,
                              const long  time,
                              const long  t1,
                              const long  x_stride0,
                              const long  y_stride0){ 

          const long  p_rec=blockDim.x*blockIdx.x+threadIdx.x+p_rec_m;

          if(p_rec<p_rec_M){

              long  posx = (long )(floorf(1.0e-1*(-o_x + rec_coordsL0(p_rec, 0))));
              long  posy = (long )(floorf(1.0e-1*(-o_y + rec_coordsL0(p_rec, 1))));
              float px = 1.0e-1F*(-o_x + rec_coordsL0(p_rec, 0)) - floorf(1.0e-1F*(-o_x + rec_coordsL0(p_rec, 0)));
              float py = 1.0e-1F*(-o_y + rec_coordsL0(p_rec, 1)) - floorf(1.0e-1F*(-o_y + rec_coordsL0(p_rec, 1)));

              float tmp=(dt*dt)*(vpL0(posx + 2, posy + 2)*vpL0(posx + 2, posy + 2))*recL0(time, p_rec);

              for (long  rrecx = 0; rrecx <= 1; rrecx += 1){
                  for (long  rrecy = 0; rrecy <= 1; rrecy += 1){
                      if (rrecx + posx >= x_m - 1 && rrecy + posy >= y_m - 1 && rrecx + posx <= x_M + 1 && rrecy + posy <= y_M + 1)
                      {
                          float r0 = tmp*(rrecx*px + (1 - rrecx)*(1 - px))*(rrecy*py + (1 - rrecy)*(1 - py));
                          atomicAdd(&vL0(t1, rrecx + posx + 4, rrecy + posy + 4), r0); 
                      }
                  }
              }

        }
    }

__global__ void third_section(float* __restrict__ grad_dev, 
                              float* __restrict__ v_dev, 
                              float* __restrict__ u_dev,
                              const float r1, 
                              const long  time, 
                              const long  t0, 
                              const long  t1, 
                              const long  t2, 
                              const long  y_stride2, 
                              const long  x_stride0, 
                              const long  y_stride0, 
                              const long  x_m, 
                              const long  x_M, 
                              const long  y_m, 
                              const long  y_M){

    const long  x=blockDim.x*blockIdx.x+threadIdx.x+x_m;
    const long  y=blockDim.y*blockIdx.y+threadIdx.y+y_m;


    if(x<x_M && y<y_M){
        gradL0(x + 1, y + 1) += -(r1*(-2.0F*vL0(t0, x + 4, y + 4)) + r1*vL0(t1, x + 4, y + 4) + r1*vL0(t2, x + 4, y + 4))*uL0(time, x + 4, y + 4);
    }

    
}


extern "C" long  Gradient(struct dataobj *__restrict__ damp_vec, struct dataobj *__restrict__ grad_vec, struct dataobj *__restrict__ rec_vec, struct dataobj *__restrict__ rec_coords_vec, struct dataobj *__restrict__ u_vec, struct dataobj *__restrict__ v_vec, struct dataobj *__restrict__ vp_vec, const long  x_M, const long  x_m, const long  y_M, const long  y_m, const float dt, const float o_x, const float o_y, const long  p_rec_M, const long  p_rec_m, const long  time_M, const long  time_m, const long  deviceid, const long  devicerm, struct profiler * timers);

long  Gradient(struct dataobj *__restrict__ damp_vec, struct dataobj *__restrict__ grad_vec, struct dataobj *__restrict__ rec_vec, struct dataobj *__restrict__ rec_coords_vec, struct dataobj *__restrict__ u_vec, struct dataobj *__restrict__ v_vec, struct dataobj *__restrict__ vp_vec, const long  x_M, const long  x_m, const long  y_M, const long  y_m, const float dt, const float o_x, const float o_y, const long  p_rec_M, const long  p_rec_m, const long  time_M, const long  time_m, const long  deviceid, const long  devicerm, struct profiler * timers)
{

  if (deviceid != -1)
  {
    hipSetDevice(deviceid);
  }


  float *damp = (float *) damp_vec->data;
  float *grad = (float *) grad_vec->data;
  float *rec = (float *) rec_vec->data;
  float *rec_coords = (float *) rec_coords_vec->data;
  float *u = (float *) u_vec->data;
  float *v = (float *) v_vec->data;
  float *vp = (float *) vp_vec->data;

  float *damp_dev;
  float *grad_dev;
  float *rec_dev;
  float *rec_coords_dev;
  float *u_dev;
  float *v_dev;
  float *vp_dev;

  hipMalloc((void**)&damp_dev,sizeof(float)*(damp_vec->size[0]*damp_vec->size[1]));
  hipMalloc((void**)&grad_dev,sizeof(float)*(grad_vec->size[0]*grad_vec->size[1]));
  hipMalloc((void**)&rec_dev,sizeof(float)*(rec_vec->size[0]*rec_vec->size[1]));
  hipMalloc((void**)&rec_coords_dev,sizeof(float)*(rec_coords_vec->size[0]*rec_coords_vec->size[1]));
  hipMalloc((void**)&u_dev,sizeof(float)*(u_vec->size[0]*u_vec->size[1]*u_vec->size[2]));
  hipMalloc((void**)&v_dev,sizeof(float)*(v_vec->size[0]*v_vec->size[1]*v_vec->size[2]));
  hipMalloc((void**)&vp_dev,sizeof(float)*(vp_vec->size[0]*vp_vec->size[1]));

  hipMemcpy(damp_dev,damp,sizeof(float)*(damp_vec->size[0]*damp_vec->size[1]),hipMemcpyHostToDevice); 
  hipMemcpy(grad_dev,grad,sizeof(float)*(grad_vec->size[0]*grad_vec->size[1]),hipMemcpyHostToDevice); 
  hipMemcpy(rec_dev,rec,sizeof(float)*(rec_vec->size[0]*rec_vec->size[1]),hipMemcpyHostToDevice); 
  hipMemcpy(rec_coords_dev,rec_coords,sizeof(float)*(rec_coords_vec->size[0]*rec_coords_vec->size[1]),hipMemcpyHostToDevice); 
  hipMemcpy(u_dev,u,sizeof(float)*(u_vec->size[0]*u_vec->size[1]*u_vec->size[2]),hipMemcpyHostToDevice); 
  hipMemcpy(v_dev,v,sizeof(float)*(v_vec->size[0]*v_vec->size[1]*v_vec->size[2]),hipMemcpyHostToDevice); 
  hipMemcpy(vp_dev,vp,sizeof(float)*(vp_vec->size[0]*vp_vec->size[1]),hipMemcpyHostToDevice); 


  const long  x_fsz0 = v_vec->size[1];
  const long  y_fsz0 = v_vec->size[2];
  const long  y_fsz1 = vp_vec->size[1];
  const long  y_fsz2 = grad_vec->size[1];
  const long  p_rec_fsz0 = rec_vec->size[1];
  const long  d_fsz0 = rec_coords_vec->size[1];

  const long  x_stride0 = x_fsz0*y_fsz0;
  const long  y_stride0 = y_fsz0;
  const long  y_stride1 = y_fsz1;
  const long  y_stride2 = y_fsz2;
  const long  p_rec_stride0 = p_rec_fsz0;
  const long  d_stride0 = d_fsz0;

  float r1 = 1.0F/(dt*dt);
  float r2 = 1.0F/dt;


  dim3 block_x_y(NTHX,NTHY);  // 16x16 threads per block
  dim3 grid_x_y((x_M - x_m + block_x_y.x) / block_x_y.x, (y_M - y_m + block_x_y.y) / block_x_y.y);


  dim3 p_rec_block(NTH);
  dim3 p_rec_grid((p_rec_M-p_rec_m+NTH)/NTH);


  for (long  time = time_M, t0 = (time)%(3), t1 = (time + 2)%(3), t2 = (time + 1)%(3); time >= time_m; time -= 1, t0 = (time)%(3), t1 = (time + 2)%(3), t2 = (time + 1)%(3))
  {
    START(section0)

      first_section<<<grid_x_y,block_x_y>>>(vp_dev,v_dev,damp_dev,r2,r1,x_m,x_M,y_m,y_M,x_stride0,y_stride0,y_stride1,t0,t1,t2);
      hipDeviceSynchronize();

    STOP(section0,timers)


    START(section1)

      
      if (rec_vec->size[0]*rec_vec->size[1] > 0 && p_rec_M - p_rec_m + 1 > 0)
      {
        second_section<<<p_rec_grid,p_rec_block>>>(vp_dev,rec_coords_dev,rec_dev,v_dev,p_rec_stride0,y_stride1,d_stride0,x_m,y_m,x_M,y_M,dt,o_x,o_y,p_rec_m,p_rec_M,time,t1,x_stride0,y_stride0);
        hipDeviceSynchronize();
      }

    STOP(section1,timers)

    START(section2)

      third_section<<<grid_x_y,block_x_y>>>(grad_dev,v_dev,u_dev,r1,time,t0,t1,t2,y_stride2,x_stride0,y_stride0,x_m,x_M,y_m,y_M);
      hipDeviceSynchronize();

    STOP(section2,timers)


  }

  printf("Timers sec1 %10.6f sec2 %10.6f sec3 %10.6f \n",timers->section0, timers->section1, timers-> section2);

  hipMemcpy(grad,grad_dev,sizeof(float)*(grad_vec->size[0]*grad_vec->size[1]),hipMemcpyDeviceToHost); 
  hipMemcpy(v,v_dev,sizeof(float)*(v_vec->size[0]*v_vec->size[1]*v_vec->size[2]),hipMemcpyDeviceToHost); 

  hipFree(grad_dev);
  hipFree(v_dev);

  hipFree(damp_dev);
  hipFree(rec_dev);
  hipFree(rec_coords_dev);
  hipFree(u_dev);
  hipFree(vp_dev);

  hipDeviceSynchronize();


  return 0;
}
